#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32

#define VECTOR_SIZE 10//1024 * 1024 * 16

__global__ void sum(double *a, double *b, double *c){
	int bx = blockDim.x * blockIdx.x + threadIdx.x;

	c[bx] = a[bx] + b[bx];
}


int main(void) {

	//host memories
	double* host_array_a = (double*)calloc(VECTOR_SIZE,sizeof(double));
	double* host_array_b = (double*)calloc(VECTOR_SIZE,sizeof(double));
	double* host_array_c = (double*)calloc(VECTOR_SIZE,sizeof(double));
	int i;
	for(i = 0; i < VECTOR_SIZE; i++){
		host_array_a[i] = sin(i);
		host_array_b[i] = cos(i);
	}

	//cuda memories
	double *device_array_a, *device_array_b, *device_array_c;
	hipMalloc(&device_array_a, VECTOR_SIZE);
	hipMalloc(&device_array_b, VECTOR_SIZE);
	hipMalloc(&device_array_c, VECTOR_SIZE);

	//copy to device
	hipMemcpy(device_array_a, host_array_a, VECTOR_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(device_array_b, host_array_b, VECTOR_SIZE, hipMemcpyHostToDevice);

	//kernel parameters
	//we know that each block has 512 threads
	//so
	long threadsPerBlock;
	long blocksPerGrid;
	if(VECTOR_SIZE < 512){
		threadsPerBlock = VECTOR_SIZE;
		blocksPerGrid = 1;
	}else{
		threadsPerBlock = 512;
		blocksPerGrid = ceil(double(VECTOR_SIZE)/double(threadsPerBlock));
	}
	printf("%ld %ld\n", threadsPerBlock, blocksPerGrid);
	sum<<<blocksPerGrid,threadsPerBlock>>>(device_array_a,device_array_b,device_array_c);

	hipMemcpy(host_array_c, device_array_c, VECTOR_SIZE, hipMemcpyDeviceToHost);

	for(i = 0; i < VECTOR_SIZE;i++){
		printf("a[%d] %lf + b[%d] %lf = %lf\n", i, host_array_a[i], i, host_array_b[i], host_array_c[i]);

	}
	printf("\n");



	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	return 0;
}
