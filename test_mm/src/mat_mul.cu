#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "checksum.h"

int gemm(double** a, double** b, double** c, long lin_a, long col_a, long lin_b,
		long col_b) {
	long i, j, k;
	if (col_a != lin_b)
		return -1;
	for (i = 0; i < lin_a; i++)
		for (j = 0; j < col_b; j++) {
			c[i][j] = 0;
			for (k = 0; k < col_a; k++)
				c[i][j] += a[i][k] * b[k][j];
		}
	return 0;
}

int gemm_1d(double* a, double* b, double* c, long lin_a, long col_a, long lin_b,
		long col_b, long col_c, long lin_c) {
	long i, j, k;
	if (col_a != lin_b)
		return -1;

	for (i = 0; i < lin_a; i++) {
		for (j = 0; j < col_b; j++) {
			long index_c = i * col_c + j;
			c[index_c] = 0;
			for (k = 0; k < col_a; k++) {
				c[index_c] += a[i * col_a + k] * b[k * col_b + j];
			}
		}
		//printf("\n");
	}
	return 0;
}







void print_mat_row_major(double *mat, long m, long n, const char *mat_name) {
	printf("ROW-MAJOR ORDER: printing %s lin %ld col %ld\n", mat_name, m, n);
	long i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++)
			printf("%ld ", (PRINT_TYPE) mat[i * n + j]);
		printf("\n");
	}
	printf("on vector 1d\n");
	for (i = 0; i < m * n; i++) {
		printf("%ld ", (PRINT_TYPE) mat[i]);
	}
	printf("\n");
}

void print_mat_collum_major(double *mat, long m, long n, const char *mat_name) {
	printf("COLLUM-MAJOR ORDER: printing %s lin %ld col %ld\n", mat_name, m, n);
	long i, j;
	for (i = 0; i < m; i++) {

		for (j = 0; j < n; j++) {
			printf("%ld ", (PRINT_TYPE) mat[j * m + i]);
		}
		printf("\n");
	}
	printf("on vector 1d\n");
	for (i = 0; i < m * n; i++) {
		printf("%ld ", (PRINT_TYPE) mat[i]);
	}
	printf("\n");

}

void fill_mat(double* t, long n) {
	long i;
	for (i = 0; i < n; i++) {
		t[i] = 1;
	}
}

void fill_mat_row_major(double *t, long m, long n) {
	long i, j;
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			t[i * n + j] = double(i);
}

void fill_mat_collum_major(double *t, long m, long n) {
	long i, j;
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			t[j * m + i] = double(i);
}
void compare(double *t, double *s, long siz) {
	long i;
	for (i = 0; i < siz; i++) {
		if (fabs(t[i]) - fabs(s[i]) > 0.0000001)
			printf("t[%ld] is diff from s[%ld] on diff %lf", i, i,
					fabs(t[i]) - fabs(s[i]));
	}
}

hipblasStatus_t dgemm_host(int width_a, int height_a, int width_b, int height_b,
		double *a, double *b, double *c) {
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const double alpha = 1;
	const double beta = 0;
	//note cublas is column primary!
	//need to transpose the order
	//checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA,
	//matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));

	hipblasStatus_t ret = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width_b,
			height_a, width_a, &alpha, b, width_b, a, width_a, &beta, c,
			width_b);

	if(HIPBLAS_STATUS_SUCCESS != ret){
		printf("pau no blas\n");
		exit(-1);
	}

	hipblasDestroy(handle);
	return ret;
}

void matrix_multiplication_abft() {
	long lin_a = 7;
	long col_a = 9;
	long lin_b = col_a;
	long col_b = 5;
	long vec_siz_a = ((lin_a + 1) * (col_a + 1));
	long vec_siz_b = ((lin_b + 1) * (col_b + 1));
	long vec_siz_c = ((lin_a + 1) * (col_b + 1));
	const long siz_a = vec_siz_a * sizeof(double);
	const long siz_b = vec_siz_b * sizeof(double);
	const long siz_c = vec_siz_b * sizeof(double);
	//host memories
	double* host_array_a = (double*) calloc(vec_siz_a, sizeof(double));
	double* host_array_b = (double*) calloc(vec_siz_b, sizeof(double));
	double* host_array_c = (double*) calloc(vec_siz_c, sizeof(double));
	double* host_array_c_temp = (double*) calloc(vec_siz_c, sizeof(double));
	fill_mat_row_major(host_array_a, lin_a + 1, col_a + 1);
	fill_mat_row_major(host_array_b, lin_b + 1, col_b + 1);

	//cuda memories
	double *device_array_a, *device_array_b, *device_array_c;
	hipMalloc(&device_array_a, siz_a);
	hipMalloc(&device_array_b, siz_b);
	hipMalloc(&device_array_c, siz_c);
	//copy to devicex_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
	hipMemcpy(device_array_a, host_array_a, siz_a, hipMemcpyHostToDevice);
	hipMemcpy(device_array_b, host_array_b, siz_b, hipMemcpyHostToDevice);



//
//	printf("blocks_abft_first %ld threads_abft_firs %ld\n", blocks_abft_first,
//			threads_abft_first);
//	printf("blocks_abft_second %ld threads_abft_second %ld\n",
//			blocks_abft_second, threads_abft_second);
	first_abraham(device_array_a, lin_a + 1, col_a + 1);
	second_abraham(device_array_b, lin_b + 1, col_b + 1);

	hipMemcpy(host_array_a, device_array_a, siz_a, hipMemcpyDeviceToHost);
	hipMemcpy(host_array_b, device_array_b, siz_b, hipMemcpyDeviceToHost);
	print_mat_row_major(host_array_a, lin_a + 1, col_a + 1, "matrix A");
	printf("\n");
	print_mat_row_major(host_array_b, lin_b + 1, col_b + 1, "matrix B");

	//hipblasStatus_t dgemm_host(int width_a, int height_a, int width_b, int height_b, double *a, double *b,	double *c)
	dgemm_host(col_a + 1, lin_a + 1, col_b + 1, lin_b + 1, device_array_a,
			device_array_b, device_array_c);

	hipMemcpy(host_array_c, device_array_c, siz_c, hipMemcpyDeviceToHost);
	print_mat_row_major(host_array_c, lin_a + 1, col_b + 1, "GPU result mat");
	int row_detected_errors_host, col_detected_errors_host;

	//abraham_check(device_array_c, (lin_a + 1), (col_b + 1));

	//hipMemcpyFromSymbol(&row_detected_errors_host, HIP_SYMBOL(row_detected_errors),sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpyFromSymbol(&col_detected_errors_host, HIP_SYMBOL(col_detected_errors),sizeof(int), hipMemcpyDeviceToHost);
	printf("Detected row errors: %d\nDetected collum errors %d\n", row_detected_errors_host, col_detected_errors_host);

	//printf("compare matrices\n");

//	free(host_array_a);
//	free(host_array_b);
//	free(host_array_c);
//	free(host_array_c_temp);
//
//	hipFree(device_array_a);
//	hipFree(device_array_b);
//	hipFree(device_array_c);
}

int main(void) {
	matrix_multiplication_abft();
	return 0;
}
//
//__global__ void mat_cpy(double *dst, double *src, long collums, long rows) {
//	long x = (blockDim.x * blockIdx.x) + threadIdx.x;
//	long y = (blockDim.y * blockIdx.y) + threadIdx.y;
//
//	long index = (collums * y) + x;
//
//	if (collums * rows > index)
//		dst[index] = src[index];
//}
//int gemm_ongpu_abft(double *a, double *b, double *c, long lin_a, long col_a,
//		long lin_b, long col_b) {
//	long i, j;
//	double acc = 0;
//	int ret = 0;
//	long col_c = col_b;
////	long lin_c = lin_a;
//	//first ABRAHAM operation
//	for (j = 0; j < col_a; j++) {
//		acc = 0;
//		for (i = 0; i < lin_a; i++)
//
//			acc += a[i * col_a + j];
//
//		a[lin_a * col_a + j] = acc;
//	}
//
//	//second ABRAHAM operation
//	for (i = 0; i < lin_b; i++) {
//		acc = 0;
//		for (j = 0; j < col_b; j++)
//			acc += b[i * (col_b + 1) + j];
//		//printf("i * col_b %ld col b %ld  acc %lf\n", i * col_b, col_b, acc);
//		b[i * (col_b + 1) + col_b] = acc;
//	}
//
//	//print_mat(a, lin_a + 1, col_a);
//	//printf("\n");
//	//print_mat(b, lin_b, col_b + 1);
//	//performs matrix multiplication
//	gemm_1d(a, b, c, lin_a + 1, col_a, lin_b, col_b + 1, col_b + 1, lin_a + 1);
//
//	//check all checksums
//	//line checksum
//	for (j = 0; j < col_a; j++) {
//		acc = 0;
//		for (i = 0; i < lin_a; i++)
//			acc += c[i * col_c + j];
//
//		if (fabs(c[lin_a * col_c + j]) - fabs(acc) >= MAX_THRESHOLD) {
////			printf(
////					"lin - position corrupted [%ld][%ld] - exp chsum %lf got chsum %lf diff - %lf\n",
////					lin_a, j, c[lin_a * col_c + j], acc,
////					c[lin_a * col_c + j] - acc);
//			ret++;
//		}
//	}
//
//	//collum checksum
//	for (i = 0; i < lin_b; i++) {
//		acc = 0;
//		for (j = 0; j < col_b; j++)
//			acc += c[i * col_c + j];
//
//		if (fabs(c[i * col_c + col_b] - acc) >= MAX_THRESHOLD) {
////			printf(
////					"collum - position corrupted [%ld][%ld] - exp chsum %lf got chsum %lf diff %lf\n",
////					i, col_b, c[i * col_c + col_b], acc,
////					c[i * col_c + col_b] - acc);
//			ret++;
//		}
//	}
//	return ret;
//
//}
//
//void matrix_multiplication_no_abft() {
//	const long siz_a = VECTOR_SIZE_A * sizeof(double);
//	const long siz_b = VECTOR_SIZE_B * sizeof(double);
//	const long siz_c = VECTOR_SIZE_C * sizeof(double);
//	//host memories
//	double* host_array_a = (double*) calloc(VECTOR_SIZE_A, sizeof(double));
//	double* host_array_b = (double*) calloc(VECTOR_SIZE_B, sizeof(double));
//	double* host_array_c = (double*) calloc(VECTOR_SIZE_C, sizeof(double));
//	double* host_array_c_temp = (double*) calloc(VECTOR_SIZE_C, sizeof(double));
//	fill_mat(host_array_a, VECTOR_SIZE_A);
//	fill_mat(host_array_b, VECTOR_SIZE_B);
//	//print_mat(host_array_a, COLLUMS_A, ROWS_A, "matrix A");
//	printf("\n");
//	//print_mat(host_array_b, COLLUMS_B, ROWS_B, "matrix B");
//	//perform host matrix multiplication
//	//	gemm_1d(host_array_a, host_array_b, host_array_c_temp, ROWS_A, COLLUMS_A,
//	//			ROWS_B, COLLUMS_B, ROWS_A, COLLUMS_B);
//	//print_mat(host_array_c_temp, COLLUMS_B, ROWS_A, "matrix C temp");
//	//cuda memories
//	double *device_array_a, *device_array_b, *device_array_c;
//	hipMalloc(&device_array_a, siz_a);
//	hipMalloc(&device_array_b, siz_b);
//	hipMalloc(&device_array_c, siz_c);
//	//copy to device
//	hipMemcpy(device_array_a, host_array_a, siz_a, hipMemcpyHostToDevice);
//	hipMemcpy(device_array_b, host_array_b, siz_b, hipMemcpyHostToDevice);
//	//kernel parameters
//	//we know that each block has 1024 threads
//	long blocks = ceil(N / float(BLOCK_SIZE));
//	long threads = ceil(N / float(blocks));
//	//2d grid
//	dim3 gridDim(blocks, blocks);
//	//threads num, 2d
//	dim3 blockDim(threads, threads);
//	mat_mult<<<gridDim, blockDim>>>(device_array_c, device_array_a,
//			device_array_b, N);
//	printf("\nblocks %ld threads %ld\n", blocks, threads);
//	hipMemcpy(host_array_c, device_array_c, siz_c, hipMemcpyDeviceToHost);
//	//print_mat(host_array_c, COLLUMS_A, ROWS_A, "GPU result mat");
//	printf("compare matrices\n");
//	//compare(host_array_c, host_array_c_temp, VECTOR_SIZE_C);
//	hipFree(device_array_a);
//	hipFree(device_array_b);
//	hipFree(device_array_c);
//	free(host_array_a);
//	free(host_array_b);
//	free(host_array_c);
//	free(host_array_c_temp);
//}
//
//__global__ void mat_mult(double *dst, double *a, double *b, long col) {
//	long i = blockIdx.y * blockDim.y + threadIdx.y;
//	long j = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (i > col || j > col)
//		return;
//
//	double acc = 0;
//	long index_dst = i * col + j;
//	long k;
//	for (k = 0; k < col; k++) {
//		acc += a[i * col + k] * b[k * col + j];
//	}
//	dst[index_dst] = acc;
//}
