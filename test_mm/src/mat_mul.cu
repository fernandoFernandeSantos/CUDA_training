#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <math.h>

inline double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
		true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}

//__device__ int row_detected_errors = 0;
//__device__ int col_detected_errors = 0;

typedef struct erro_return {
	long* row_detected_errors;
	long* col_detected_errors;

	long* row_detected_errors_gpu;
	long* col_detected_errors_gpu;

	int error_status;
} ErrorReturn;

__device__ ErrorReturn err_count;



/* Finds the sum of all elements in the row excluding the element at eRow and the checksum element */
__global__ float excl_row_sum(float *mat, long rows, long cols, long error_row, long error_col) {
    long i = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (matrix == NULL)
        errExit("Matrix is NULL. Cannot sum.");
    if (row >= rows) {
        errExit("Error row exceeds the number of rows.");
    }
    else if (col >= cols) {
        errExit("Error column exceeds the number of columns.");
    }
    for (i = 0; i < cols - 1; i++) {
        /* if i is not the trouble column */
        if (i != col)
            sum += matrix[row][i];
    }
    return sum;
}

/* Finds the sum of all elements in the col excluding the element at eRow and the checksum element */
int excl_col_sum(float *mat, long rows, long cols, long error_row) {
    long j = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (mat == NULL || error_row > rows)
    	atomicAdd(&err_count.error_status, 1);
    long i;
    for (i = 0; i < rows - 1; i++) {
        /* if j is not the trouble row */
        if (i != error_row){
        	long index = get_index(i, j, cols);
        	sum += mat[index];
        }
    }
    return sum;
}



#define BLOCK_SIZE 1024

#define DIV_VALUE 1e5

#define MAX_THRESHOLD  0.05
#define PRINT_TYPE double

__device__ inline long get_index(long i, long j, long n){
	return i * n + j;
}

__global__ void check_col(float *mat, long rows, long cols) {
	long i = blockIdx.x * blockDim.x + threadIdx.x;
	long b_index = i * cols + cols - 1;
	if (cols == 1 || b_index > (rows * cols))
		return;

	long k;
	double acc = 0;
	//must be less one
	for (k = 0; k < cols - 1; k++) {
		acc += (mat[i * cols + k] / DIV_VALUE);
	}

	//printf("b_index %ld acc %lf \n", b_index, acc);
	float diff = fabs(mat[b_index] - acc);
	if (diff >= MAX_THRESHOLD) {
		atomicAdd(&col_detected_errors, 1);
//		printf("passou no col mat[%ld] = %ld diff %ld calc %ld i %ld\n",
//				b_index, (long) mat[b_index], (long) diff, (long) acc, i);
	}
	//__syncthreads();
}

__global__ void check_row(float *mat, long rows, long cols) {
	long j = blockIdx.x * blockDim.x + threadIdx.x;
	long a_index = (rows - 1) * cols + j;
	if (rows == 1 || a_index > (rows * cols))
		return;

	long k;
	double acc = 0;
	//must be less one
	for (k = 0; k < rows - 1; k++) {
		acc += (mat[k * cols + j] / DIV_VALUE);
	}
	//printf("a_index %ld acc %lf \n", rows_a * cols_a + j, acc);

	float diff = fabs(mat[a_index] - acc);
	if (diff >= MAX_THRESHOLD) {
		atomicAdd(&row_detected_errors, 1);
//		printf("passou no row mat[%ld] = %lf diff %lf calc %lf i value %ld\n",
//				a_index, mat[a_index - 1], diff, acc, j);
	}
	//__syncthreads();
}


//since dgemm is optimized for square matrices I'm going to use
//first ABRAHAM operation
//	for (j = 0; j < col_a; j++) {
//		acc = 0;
//		for (i = 0; i < lin_a; i++)
//
//			acc += a[i * col_a + j];
//
//        a[lin_a * col_a + j] = acc;
//	}
//rows_b MUST BE THE SAME OF cols_a
__global__ void first_abraham_op(float *a, long rows_a, long cols_a) {
	long j = blockIdx.x * blockDim.x + threadIdx.x;
	long a_index = get_index((rows_a - 1), j, cols_a);

	if (rows_a == 1 || a_index > (rows_a * cols_a))
		return;

	long k;
	double acc = 0;
	for (k = 0; k < rows_a - 1; k++) {
		long index = get_index(k, j, cols_a);
		acc += (a[index] / DIV_VALUE);
	}

	a[a_index] = acc;
}

/**
 * 	for (i = 0; i < lin_b; i++) {
 acc = 0;
 for (j = 0; j < col_b; j++)
 acc += b[i * (col_b + 1) + j];
 //printf("i * col_b %ld col b %ld  acc %lf\n", i * col_b, col_b, acc);
 b[i * (col_b + 1) + col_b] = acc;
 }
 */
__global__ void second_abraham_op(float *b, long rows_b, long cols_b) {
	long i = blockIdx.x * blockDim.x + threadIdx.x;
	long b_index = get_index(i, cols_b - 1, cols_b);
	if (rows_b == 1 || b_index > (rows_b * cols_b))
		return;

	long k;
	double acc = 0;
	for (k = 0; k < cols_b - 1; k++) {
		long index = get_index(i, k, cols_b);
		acc += (b[index] / DIV_VALUE);
	}
	b[b_index] = acc;
}

void check_checksums_from_host(float *c, long rows_c, long cols_c) {
	long blocks = ceil(float(cols_c) / float(BLOCK_SIZE));
	long threads = ceil(float(cols_c) / float(blocks));
	check_row<<<blocks, threads>>>(c, rows_c, cols_c);
	blocks = ceil(float(rows_c) / float(BLOCK_SIZE));
	threads = ceil(float(rows_c) / float(blocks));
	check_col<<<blocks, threads>>>(c, rows_c, cols_c);
}


void calc_checksums_from_host(float *a, float *b, long rows_a, long cols_a,
		long rows_b, long cols_b) {
	//1d grid for abft operations
//	long *temp;
//	long temp_host[cols_a];
//	hipMalloc(&temp, cols_a * sizeof(long));

	long blocks = ceil(float(cols_a) / float(BLOCK_SIZE));
	long threads = ceil(float(cols_a) / float(blocks));

	first_abraham_op<<<blocks, threads>>>(a, rows_a, cols_a);

//	hipMemcpy(temp_host, temp, cols_a * sizeof(long), hipMemcpyDeviceToHost);


	printf("first blocks %ld threads %ld\n", blocks, threads);
	//second
	blocks = ceil(float(rows_b) / float(BLOCK_SIZE));
	threads = ceil(float(rows_b) / float(blocks));
	second_abraham_op<<<blocks, threads>>>(b, rows_b, cols_b);
	printf("second blocks %ld threads %ld\n", blocks, threads);
}

void print_mat_row_major(float *mat, long m, long n, const char *mat_name) {
	if (m * n > 5000)
		return;
	printf("ROW-MAJOR ORDER: printing %s lin %ld col %ld\n", mat_name, m, n);
	long i, j;
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++)
			printf("%lf ", (PRINT_TYPE) mat[i * n + j]);
		printf("\n");
	}
//	printf("on vector 1d\n");
//	for (i = 0; i < m * n; i++) {
//		printf("%ld ", (PRINT_TYPE) mat[i]);
//	}
	printf("\n");
}

void fill_mat(float* t, long n) {
	long i;
	for (i = 0; i < n; i++) {
		t[i] = 1;
	}
}

void fill_mat_row_major(float *t, long m, long n) {
	long i, j;
	for (i = 0; i < m; i++)
		for (j = 0; j < n; j++)
			t[i * n + j] = 1; //((rand() % 15) / 3.14578);
}

void compare(float *t, float *s, long siz) {
	long i;
	for (i = 0; i < siz; i++) {
		if (fabs(t[i]) - fabs(s[i]) > 0.0000001)
			printf("t[%ld] is diff from s[%ld] on diff %lf", i, i,
					fabs(t[i]) - fabs(s[i]));
	}
}

/**
 Matrix multiplication: C = A * B.
 Host code.

 This sample implements matrix multiplication as described in Chapter 3
 of the programming guide and uses the CUBLAS library to demonstrate
 the best performance.

 SOME PRECAUTIONS:
 IF WE WANT TO CALCULATE ROW-MAJOR MATRIX MULTIPLY C = A * B,
 WE JUST NEED CALL CUBLAS API IN A REVERSE ORDER: cublasSegemm(B, A)!
 The reason is explained as follows:

 CUBLAS library uses column-major storage, but C/C++ use row-major storage.
 When passing the matrix pointer to CUBLAS, the memory layout alters from
 row-major to column-major, which is equivalent to an implicit transpose.

 In the case of row-major C/C++ matrix A, B, and a simple matrix multiplication
 C = A * B, we can't use the input order like hipblasSgemm(A, B)  because of
 implicit transpose. The actual result of cublasSegemm(A, B) is A(T) * B(T).
 If col(A(T)) != row(B(T)), equal to row(A) != col(B), A(T) and B(T) are not
 multipliable. Moreover, even if A(T) and B(T) are multipliable, the result C
 is a column-based cublas matrix, which means C(T) in C/C++, we need extra
 transpose code to convert it to a row-based C/C++ matrix.

 To solve the problem, let's consider our desired result C, a row-major matrix.
 In cublas format, it is C(T) actually (because of the implicit transpose).
 C = A * B, so C(T) = (A * B) (T) = B(T) * A(T). Cublas matrice B(T) and A(T)
 happen to be C/C++ matrice B and A (still because of the implicit transpose)!
 We don't need extra transpose code, we only need alter the input order!

 CUBLAS provides high-performance matrix multiplication.
 See also:
 V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

hipblasStatus_t dgemm_host(int width_a, int height_a, int width_b, int height_b,
		float *a, float *b, float *c) {
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1;
	const float beta = 0;
	//note cublas is column primary!
	//need to transpose the order
//	m input	number of rows of matrix op(A) and C.
//	n input	number of columns of matrix op(B) and C.
//	k input number of columns of op(A) and rows of op(B).
//  lda == m
//  ldb == k
//  ldc == m
//checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB,
	//d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
	int lda = width_a;
	int ldb = width_b;
	int ldc = width_b;
	hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width_b,
			height_a, width_a, &alpha, b, ldb, a, lda, &beta, c, ldc);

	if (HIPBLAS_STATUS_SUCCESS != ret) {
		printf("pau no blas\n");
		exit(-1);
	}

	hipblasDestroy(handle);
	return ret;
}

void matrix_multiplication_abft() {
	long lin_a = 2005;//96;
	long col_a = 1255;//48;
	long lin_b = col_a;//48;
	long col_b = 1402;//92;
	long vec_siz_a = ((lin_a) * (col_a));
	long vec_siz_b = ((lin_b) * (col_b));
	long vec_siz_c = ((lin_a) * (col_b));
	const long siz_a = vec_siz_a * sizeof(float);
	const long siz_b = vec_siz_b * sizeof(float);
	const long siz_c = vec_siz_c * sizeof(float);
	//host memories
	float* host_array_a = (float*) calloc(vec_siz_a, sizeof(float));
	float* host_array_b = (float*) calloc(vec_siz_b, sizeof(float));
	float* host_array_c = (float*) calloc(vec_siz_c, sizeof(float));
//	float* host_array_c_temp = (float*) calloc(vec_siz_c, sizeof(float));
	fill_mat_row_major(host_array_a, lin_a, col_a);
	fill_mat_row_major(host_array_b, lin_b, col_b);

	//cuda memories
	float *device_array_a, *device_array_b, *device_array_c;
	hipMalloc(&device_array_a, siz_a);
	hipMalloc(&device_array_b, siz_b);
	hipMalloc(&device_array_c, siz_c);
	//copy to devicex_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
	hipMemcpy(device_array_a, host_array_a, siz_a, hipMemcpyHostToDevice);
	hipMemcpy(device_array_b, host_array_b, siz_b, hipMemcpyHostToDevice);

	double time_from_host = mysecond();
	calc_checksums_from_host(device_array_a, device_array_b, lin_a, col_a,
			lin_b, col_b);
	printf("Calc checksums time calling from host %lf\n",
			mysecond() - time_from_host);

	hipMemcpy(host_array_a, device_array_a, siz_a, hipMemcpyDeviceToHost);
	hipMemcpy(host_array_b, device_array_b, siz_b, hipMemcpyDeviceToHost);
	print_mat_row_major(host_array_a, lin_a, col_a, "matrix A");

	print_mat_row_major(host_array_b, lin_b, col_b, "matrix B");

	dgemm_host(col_a, lin_a, col_b, lin_b, device_array_a, device_array_b,
			device_array_c);

	hipMemcpy(host_array_c, device_array_c, siz_c, hipMemcpyDeviceToHost);
	print_mat_row_major(host_array_c, lin_a, col_b, "GPU result mat");
	int row_detected_errors_host = 0, col_detected_errors_host = 0;

	time_from_host = mysecond();
	check_checksums_from_host(device_array_c, (lin_a), (col_b));
	printf("Final check time calling from host %lf\n",
			mysecond() - time_from_host);

	hipMemcpyFromSymbol(&row_detected_errors_host, HIP_SYMBOL(row_detected_errors),
			sizeof(int));
	hipMemcpyFromSymbol(&col_detected_errors_host, HIP_SYMBOL(col_detected_errors),
			sizeof(int));
	printf("Detected row errors: %d\nDetected collum errors %d\n",
			row_detected_errors_host, col_detected_errors_host);
	printf("\n");

	gpuErrchk(hipDeviceSynchronize());
	free(host_array_a);
	free(host_array_b);
	free(host_array_c);

	hipFree(device_array_a);
	hipFree(device_array_b);
	hipFree(device_array_c);
}

int main(void) {
	matrix_multiplication_abft();
	return 0;
}
