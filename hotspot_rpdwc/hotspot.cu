
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#include <vector>
#include <iostream>

#ifdef RD_WG_SIZE_0_0                                                            
#define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)                                                      
#define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)                                                        
#define BLOCK_SIZE RD_WG_SIZE
#else                                                                                    
#define BLOCK_SIZE 16
#endif                                                                                   

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

void fatal(char *s) {
	fprintf(stderr, "error: %s\n", s);

}

void writeoutput(float *vect, int grid_rows, int grid_cols, char *file) {

	int i, j, index = 0;
	FILE *fp;
	char str[STR_SIZE];

	if ((fp = fopen(file, "w")) == 0)
		printf("The file was not opened\n");

	for (i = 0; i < grid_rows; i++)
		for (j = 0; j < grid_cols; j++) {

			sprintf(str, "%d\t%g\n", index, vect[i * grid_cols + j]);
			fputs(str, fp);
			index++;
		}

	fclose(fp);
}

template<typename double_t, typename single_t>
void compareOutputHost(std::vector<double_t> &vectDouble,
		std::vector<single_t> &vectSingle) {
	single_t max_relative = -99999;
	single_t min_relative = 99999;

	for (int i = 0; i < vectDouble.size(); i++) {
		auto dt = vectDouble[i];
		auto st = vectSingle[i];
		auto diff = (st - single_t(dt)) / st;
		max_relative = std::max(max_relative, diff);
		min_relative = std::min(min_relative, diff);
	}

	std::cout << "Max relative error on host " << max_relative << std::endl;
	std::cout << "Min relative error on host " << min_relative << std::endl;
}

template<typename double_t, typename single_t> __global__
void compareOutputGPU(double_t *vectDouble, single_t *vectSingle) {
	auto index = threadIdx.x;
	auto dt = vectDouble[index];
	auto st = vectSingle[index];

}

void readinput(double *vectDouble, float *vect, int grid_rows, int grid_cols,
		char *file) {

	int i, j;
	FILE *fp;
	char str[STR_SIZE];
	float val;

	if ((fp = fopen(file, "r")) == 0)
		printf("The file was not opened\n");

	for (i = 0; i <= grid_rows - 1; i++)
		for (j = 0; j <= grid_cols - 1; j++) {
			fgets(str, STR_SIZE, fp);
			if (feof(fp))
				fatal("not enough lines in file");
			//if ((sscanf(str, "%d%f", &index, &val) != 2) || (index != ((i-1)*(grid_cols-2)+j-1)))
			if ((sscanf(str, "%f", &val) != 1))
				fatal("invalid file format");
			vect[i * grid_cols + j] = val;
			vectDouble[i * grid_cols + j] = val;
		}

	fclose(fp);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

template<typename real_t>
__global__ void calculate_temp(int iteration,  //number of iteration
		real_t *power,   //power input
		real_t *temp_src,    //temperature input/output
		real_t *temp_dst,    //temperature input/output
		int grid_cols,  //Col of grid
		int grid_rows,  //Row of grid
		int border_cols,  // border offset
		int border_rows,  // border offset
		real_t Cap,      //Capacitance
		real_t Rx, real_t Ry, real_t Rz, real_t step, real_t time_elapsed) {

	__shared__ real_t temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ real_t power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ real_t temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

	real_t amb_temp = real_t(80.0f);
	real_t step_div_Cap;
	real_t Rx_1, Ry_1, Rz_1;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	step_div_Cap = step / Cap;

	Rx_1 = 1 / Rx;
	Ry_1 = 1 / Ry;
	Rz_1 = 1 / Rz;

	// each block finally computes result for a small block
	// after N iterations.
	// it is the non-overlapping small blocks that cover
	// all the input data

	// calculate the small block size
	int small_block_rows = BLOCK_SIZE - iteration * 2;        //EXPAND_RATE
	int small_block_cols = BLOCK_SIZE - iteration * 2;        //EXPAND_RATE

	// calculate the boundary for the block according to
	// the boundary of its small block
	int blkY = small_block_rows * by - border_rows;
	int blkX = small_block_cols * bx - border_cols;
	int blkYmax = blkY + BLOCK_SIZE - 1;
	int blkXmax = blkX + BLOCK_SIZE - 1;

	// calculate the global thread coordination
	int yidx = blkY + ty;
	int xidx = blkX + tx;

	// load data if it is within the valid input range
	int loadYidx = yidx, loadXidx = xidx;
	int index = grid_cols * loadYidx + loadXidx;

	if (IN_RANGE(loadYidx, 0,
			grid_rows - 1) && IN_RANGE(loadXidx, 0, grid_cols - 1)) {
		temp_on_cuda[ty][tx] = temp_src[index]; // Load the temperature data from global memory to shared memory
		power_on_cuda[ty][tx] = power[index]; // Load the power data from global memory to shared memory
	}
	__syncthreads();

	// effective range within this block that falls within
	// the valid range of the input data
	// used to rule out computation outside the boundary.
	int validYmin = (blkY < 0) ? -blkY : 0;
	int validYmax = (blkYmax > grid_rows - 1) ?
	BLOCK_SIZE - 1 - (blkYmax - grid_rows + 1) :
												BLOCK_SIZE - 1;
	int validXmin = (blkX < 0) ? -blkX : 0;
	int validXmax = (blkXmax > grid_cols - 1) ?
	BLOCK_SIZE - 1 - (blkXmax - grid_cols + 1) :
												BLOCK_SIZE - 1;

	int N = ty - 1;
	int S = ty + 1;
	int W = tx - 1;
	int E = tx + 1;

	N = (N < validYmin) ? validYmin : N;
	S = (S > validYmax) ? validYmax : S;
	W = (W < validXmin) ? validXmin : W;
	E = (E > validXmax) ? validXmax : E;

	bool computed;
	for (int i = 0; i < iteration; i++) {
		computed = false;
		if ( IN_RANGE(tx, i + 1, BLOCK_SIZE-i-2) &&
		IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&
		IN_RANGE(tx, validXmin, validXmax) &&
		IN_RANGE(ty, validYmin, validYmax)) {
			computed = true;
			temp_t[ty][tx] = temp_on_cuda[ty][tx]
					+ step_div_Cap
							* (power_on_cuda[ty][tx]
									+ (temp_on_cuda[S][tx] + temp_on_cuda[N][tx]
											- 2.0 * temp_on_cuda[ty][tx]) * Ry_1
									+ (temp_on_cuda[ty][E] + temp_on_cuda[ty][W]
											- 2.0 * temp_on_cuda[ty][tx]) * Rx_1
									+ (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);

		}
		__syncthreads();
		if (i == iteration - 1)
			break;
		if (computed)	 //Assign the computation range
			temp_on_cuda[ty][tx] = temp_t[ty][tx];
		__syncthreads();
	}

	// update the global memory
	// after the last iteration, only threads coordinated within the
	// small block perform the calculation and switch on ``computed''
	if (computed) {
		temp_dst[index] = temp_t[ty][tx];
	}
}

/*
 compute N time steps
 */
template<typename real_t>
int compute_tran_temp(real_t *MatrixPower, real_t *MatrixTemp[2], int col,
		int row, int total_iterations, int num_iterations, int blockCols,
		int blockRows, int borderCols, int borderRows) {
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(blockCols, blockRows);

	real_t grid_height = chip_height / row;
	real_t grid_width = chip_width / col;

	real_t Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	real_t Rx = grid_width / (real_t(2.0) * K_SI * t_chip * grid_height);
	real_t Ry = grid_height / (real_t(2.0) * K_SI * t_chip * grid_width);
	real_t Rz = t_chip / (K_SI * grid_height * grid_width);

	real_t max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	real_t step = PRECISION / max_slope;
	real_t t;
	real_t time_elapsed = real_t(0.001);

	int src = 1, dst = 0;

	for (t = 0; t < total_iterations; t += num_iterations) {
		int temp = src;
		src = dst;
		dst = temp;
		calculate_temp<<<dimGrid, dimBlock>>>(
				MIN(num_iterations, total_iterations - t), MatrixPower,
				MatrixTemp[src], MatrixTemp[dst], col, row, borderCols,
				borderRows, Cap, Rx, Ry, Rz, step, time_elapsed);
	}
	return dst;
}

void usage(int argc, char **argv) {
	fprintf(stderr,
			"Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n",
			argv[0]);
	fprintf(stderr,
			"\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
	fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr,
			"\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr,
			"\t<power_file> - name of the file containing the dissipated power values of each cell\n");
	fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char** argv) {
	printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

	run(argc, argv);

	return EXIT_SUCCESS;
}

void run(int argc, char** argv) {
	int size;
	int grid_rows, grid_cols;
//	float *FilesavingTempFloat, *FilesavingPowerFloat, *MatrixOutFloat;
//	double *FilesavingTempDouble, *FilesavingPowerDouble, *MatrixOutDouble;

	std::vector<float> FilesavingTempFloat, FilesavingPowerFloat,
			MatrixOutFloat;
	std::vector<double> FilesavingTempDouble, FilesavingPowerDouble,
			MatrixOutDouble;

	char *tfile, *pfile, *ofile;

	int total_iterations = 60;
	int pyramid_height = 1; // number of iterations

	if (argc != 7)
		usage(argc, argv);
	if ((grid_rows = atoi(argv[1])) <= 0 || (grid_cols = atoi(argv[1])) <= 0
			|| (pyramid_height = atoi(argv[2])) <= 0 || (total_iterations =
					atoi(argv[3])) <= 0)
		usage(argc, argv);

	tfile = argv[4];
	pfile = argv[5];
	ofile = argv[6];

	size = grid_rows * grid_cols;

	/* --------------- pyramid parameters --------------- */
# define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
	int borderCols = (pyramid_height) * EXPAND_RATE / 2;
	int borderRows = (pyramid_height) * EXPAND_RATE / 2;
	int smallBlockCol = BLOCK_SIZE - (pyramid_height) * EXPAND_RATE;
	int smallBlockRow = BLOCK_SIZE - (pyramid_height) * EXPAND_RATE;
	int blockCols = grid_cols / smallBlockCol
			+ ((grid_cols % smallBlockCol == 0) ? 0 : 1);
	int blockRows = grid_rows / smallBlockRow
			+ ((grid_rows % smallBlockRow == 0) ? 0 : 1);

//	FilesavingTempFloat = (float *) malloc(size * sizeof(float));
//	FilesavingPowerFloat = (float *) malloc(size * sizeof(float));
//	MatrixOutFloat = (float *) calloc(size, sizeof(float));
//	if (!FilesavingPowerFloat || !FilesavingTempFloat || !MatrixOutFloat)
//		fatal("unable to allocate memory");

	FilesavingTempFloat.resize(size);
	FilesavingPowerFloat.resize(size);
	MatrixOutFloat.resize(size);

	FilesavingTempDouble.resize(size);
	FilesavingPowerDouble.resize(size);
	MatrixOutDouble.resize(size);

	printf(
			"pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",
			pyramid_height, grid_cols, grid_rows, borderCols, borderRows,
			blockCols, blockRows, smallBlockCol, smallBlockRow);

	//READ FOR FLOAT and convert to DOUBLE
	readinput(FilesavingTempDouble.data(), FilesavingTempFloat.data(),
			grid_rows, grid_cols, tfile);
	readinput(FilesavingPowerDouble.data(), FilesavingPowerFloat.data(),
			grid_rows, grid_cols, pfile);

	float *MatrixTemp[2], *MatrixPower;
	hipMalloc((void**) &MatrixTemp[0], sizeof(float) * size);
	hipMalloc((void**) &MatrixTemp[1], sizeof(float) * size);
	hipMemcpy(MatrixTemp[0], FilesavingTempFloat.data(), sizeof(float) * size,
			hipMemcpyHostToDevice);

	hipMalloc((void**) &MatrixPower, sizeof(float) * size);
	hipMemcpy(MatrixPower, FilesavingPowerFloat.data(), sizeof(float) * size,
			hipMemcpyHostToDevice);

	// -------------------------------------------------------------------------

	double *MatrixTempDouble[2], *MatrixPowerDouble;
	hipMalloc((void**) &MatrixTempDouble[0], sizeof(double) * size);
	hipMalloc((void**) &MatrixTempDouble[1], sizeof(double) * size);
	hipMemcpy(MatrixTempDouble[0], FilesavingTempDouble.data(),
			sizeof(double) * size, hipMemcpyHostToDevice);

	hipMalloc((void**) &MatrixPowerDouble, sizeof(double) * size);
	hipMemcpy(MatrixPowerDouble, FilesavingPowerDouble.data(),
			sizeof(double) * size, hipMemcpyHostToDevice);
	// -------------------------------------------------------------------------

	printf("Start computing the transient temperature\n");
	int ret = compute_tran_temp(MatrixPower, MatrixTemp, grid_cols, grid_rows,
			total_iterations, pyramid_height, blockCols, blockRows, borderCols,
			borderRows);

	auto retDouble = compute_tran_temp(MatrixPowerDouble, MatrixTempDouble,
			grid_cols, grid_rows, total_iterations, pyramid_height, blockCols,
			blockRows, borderCols, borderRows);
	hipDeviceSynchronize();

	printf("Ending simulation\n");
	hipMemcpy(MatrixOutFloat.data(), MatrixTemp[ret], sizeof(float) * size,
			hipMemcpyDeviceToHost);

	hipMemcpy(MatrixOutDouble.data(), MatrixTempDouble[retDouble], sizeof(double) * size,
			hipMemcpyDeviceToHost);

//	writeoutput(MatrixOutFloat, grid_rows, grid_cols, ofile);
	compareOutputHost(MatrixOutDouble, MatrixOutFloat);

	hipFree(MatrixPower);
	hipFree(MatrixTemp[0]);
	hipFree(MatrixTemp[1]);

	hipFree(MatrixPowerDouble);
	hipFree(MatrixTempDouble[0]);
	hipFree(MatrixTempDouble[1]);

//	free(MatrixOutFloat);
}
